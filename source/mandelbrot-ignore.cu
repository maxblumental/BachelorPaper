#include "hip/hip_runtime.h"
/** @file histo-global.cu histogram with global memory atomics */

#include <assert.h>
#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>
using namespace std;
/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
        }

#define cucheck_dev(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	printf("%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	assert(0);																												\
	}\
	}

//Threads per block
#define SUBDIV 32

/** time spent in device */
double gpu_time = 0;

/** a useful function to compute the number of threads */
__host__ __device__ int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell);

/** save the dwell into a PNG file 
		@remarks: code to save PNG file taken from here 
		  (error handling is removed):
		http://www.labbookpages.co.uk/software/imgProc/libPNG.html
 */
void save_image(const char *filename, int *dwells, int w, int h) {
	png_bytep row;
	
	FILE *fp = fopen(filename, "wb");
	png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, 0, 0, 0);
	png_infop info_ptr = png_create_info_struct(png_ptr);
	// exception handling
	setjmp(png_jmpbuf(png_ptr));
	png_init_io(png_ptr, fp);
	// write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, w, h,
                     8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
	             PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
	// set title
	png_text title_text;
	title_text.compression = PNG_TEXT_COMPRESSION_NONE;
	title_text.key = "Title";
	title_text.text = "Mandelbrot set, per-pixel";
	png_set_text(png_ptr, info_ptr, &title_text, 1);
	png_write_info(png_ptr, info_ptr);

	// write image data
	row = (png_bytep) malloc(3 * w * sizeof(png_byte));
	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			int r, g, b;
			dwell_color(&r, &g, &b, dwells[y * w + x]);
			row[3 * x + 0] = (png_byte)r;
			row[3 * x + 1] = (png_byte)g;
			row[3 * x + 2] = (png_byte)b;
		}
		png_write_row(png_ptr, row);
	}
	png_write_end(png_ptr, NULL);

  fclose(fp);
  png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
  png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
  free(row);
}  // save_image


/** a simple complex type */
struct complex {
	__host__ __device__ complex(float re, float im = 0) {
		this->re = re;
		this->im = im;
	}
	/** real and imaginary part */
	float re, im;
}; // struct complex

// operator overloads for complex numbers
inline __host__ __device__ complex operator+
(const complex &a, const complex &b) {
	return complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ complex operator-
(const complex &a) { return complex(-a.re, -a.im); }
inline __host__ __device__ complex operator-
(const complex &a, const complex &b) {
	return complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ complex operator*
(const complex &a, const complex &b) {
	return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const complex &a) {
	return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ complex operator/
(const complex &a, const complex &b) {
	float invabs2 = 1 / abs2(b);
	return complex((a.re * b.re + a.im * b.im) * invabs2,
								 (a.im * b.re - b.im * a.re) * invabs2);
}  // operator/

#define MAX_DWELL 256
#define BS 256
/** computes the dwell for a single pixel */
__device__ int pixel_dwell
(int w, int h, complex cmin, complex cmax, int x, int y) {
	complex dc = cmax - cmin;
	float fx = (float)x / w, fy = (float)y / h;
	complex c = cmin + complex(fx * dc.re, fy * dc.im);
	int dwell = 0;
	complex z = c;
	while(dwell < MAX_DWELL && abs2(z) < 2 * 2) {
		z = z * z + c;
		dwell++;
	}
	return dwell;
}  // pixel_dwell

__global__ void mandelbrot_k
(int *dwells, int w, int h, complex cmin, complex cmax) {
	// complex value to start iteration (c)
	int y = threadIdx.x + blockIdx.x * blockDim.x;

        for (int x = 0; x < w; x++)
          dwells[y * w + x] = pixel_dwell(w, h, cmin, cmax, x, y); 
}  // mandelbrot_k

/** gets the color, given the dwell (on host) */
#define CUT_DWELL (MAX_DWELL / 4)
void dwell_color(int *r, int *g, int *b, int dwell) {
	// black for the Mandelbrot set
	if(dwell >= MAX_DWELL) {
		*r = *g = *b = 0;
	} else {
		// cut at zero
		if(dwell < 0)
			dwell = 0;
		if(dwell <= CUT_DWELL) {
			// from black to blue the first half
			*r = *g = 0;
			*b = 128 + dwell * 127 / (CUT_DWELL);
		} else {
			// from blue to white for the second half
			*b = 255;
			*r = *g = (dwell - CUT_DWELL) * 255 / (MAX_DWELL - CUT_DWELL);
		}
	}
}  // dwell_color

int main(int argc, char **argv) {
        if (argc != 2)
          {
            fprintf(stderr, "Provide image size, please.\n");
            return 0;
          }

	// allocate memory
	int w = atoi(argv[1])*1024, h = atoi(argv[1])*1024;
	size_t dwell_sz = w * h * sizeof(int);
	int *h_dwells, *d_dwells;
	cucheck(hipMalloc((void**)&d_dwells, dwell_sz));
	h_dwells = (int*)malloc(dwell_sz);

	// compute the dwells, copy them back
	double t1 = omp_get_wtime();
	mandelbrot_k<<<divup(h, SUBDIV), SUBDIV>>>
		(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1));
	cucheck(hipDeviceSynchronize());
	double t2 = omp_get_wtime();
	cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
	gpu_time = t2 - t1;
	
	// save the image to PNG 
	save_image("mandelbrot-set-ignore.png", h_dwells, w, h);

	// print performance
            cout << gpu_time << ' ' << w*h/(1000000*gpu_time) << endl;
	// free data
	hipFree(d_dwells);
	free(h_dwells);
	return 0;
}  // main
